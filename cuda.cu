#include "hip/hip_runtime.h"
// nvcc cuda.cu -o cuda.out -gencode=arch=compute_75,code=compute_75
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <string>
#include <hip/hip_runtime.h>
#include <ctime>
#include "config.h"

#define BLOCK_SIZE 128



__global__ void wakeGPU(int reps);
__global__ void floydWarshallKernel(int k, int *G, int N);

void floydWarshall(int *matrix, const int N, int bsize);
void populateMatrix(int *matrix, int n, int density);
void showDistances(int matrix[], int n);



int main(int argc, char* argv[])
{
	int n, density, bsize;

	if(argc <= 3)
	{
		n = DEFAULT;
		density = 100;
		bsize = BLOCK_SIZE;
	}
	else
	{
		n = atoi(argv[1]);
		density = atoi(argv[2]);
		bsize = atoi(argv[3]);
	}

	
	const int size = n * n * sizeof(int);

	printf("%d %d %d", n, density, bsize);
		
	int* matrix = (int *) malloc(size);

	populateMatrix(matrix, n, density);

	printf("*** Adjacency matrix:\n");
	showDistances(matrix, n);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	wakeGPU<<<1, bsize>>>(32);

	hipEventRecord(start);

	floydWarshall(matrix, n, bsize);

	hipEventRecord(stop);


	hipEventSynchronize(stop);
	float accum = 0;
	hipEventElapsedTime(&accum, start, stop);

	printf("*** The solution is:\n");
	showDistances(matrix, n);

	printf("[GPGPU] Total elapsed time %f ms\n", accum);	
	
	free(matrix);
	
	return 0;
}



__global__ void wakeGPU(int reps)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= reps)
	{
		return;
	}
}

__global__ void floydWarshallKernel(int u, int *G, int n)
{
	int v1 = blockDim.y * blockIdx.y + threadIdx.y;
	int v2 = blockDim.x * blockIdx.x + threadIdx.x;

	if (v1 < n && v2 < n) 
	{
		int newPath = G[v1 * n + u] + G[u * n + v2];
		int oldPath = G[v1 * n + v2];
		if (oldPath > newPath)
		{
			G[v1 * n + v2] = newPath;		
		}
	}
}


void floydWarshall(int *matrix, const int n, int bsize)
{
	int *deviceMatrix;
	int size = n * n * sizeof(int);

	hipMalloc((int **) &deviceMatrix, size);	
	hipMemcpy(deviceMatrix, matrix, size, hipMemcpyHostToDevice);
	

	dim3 dimGrid((n + bsize - 1) / bsize, n);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(floydWarshallKernel), hipFuncCachePreferL1);
	for(int k = 0; k < n; k++)
	{
		floydWarshallKernel<<<dimGrid, bsize>>>(k, deviceMatrix, n);
	}
	hipDeviceSynchronize();

	hipMemcpy(matrix, deviceMatrix, size, hipMemcpyDeviceToHost);

	hipFree(deviceMatrix);
}

void showDistances(int matrix[], int n)
{
	if(PRINTABLE)
	{
		int i, j;
		printf("     ");
		for(i = 0; i < n; i++)
		{
			printf("[%d]  ", i);
		}
		printf("\n");
		for(i = 0; i < n; i++) {
			printf("[%d]", i);
			for(j = 0; j < n; j++)
			{
				if(matrix[i * n + j] == INF)
				{
					printf("  inf");
				}
				else
				{
					printf("%5d", matrix[i * n + j]);
				}
				
			}
			printf("\n");
		}
		printf("\n");
	}
}

void populateMatrix(int *matrix, int n, int density)
{
	uint i, j, value;
	srand(42);

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++){
			if(i == j)
			{
				matrix[i*n+j] = 0;
			}
			else
			{
				value = 1 + rand() % MAX;
				if(value > density)
				{
					matrix[i*n+j] = INF;
				}
				else
				{
					matrix[i*n+j] = value;
				}
			}

		}
	}
}